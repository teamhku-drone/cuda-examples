
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define N 512 
  
__global__ void add(int *a, int *b, int *c) {
  c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void initialize(int *a, int n) {
  for (int i=0; i<n; ++i) {
    a[i] = i;
  } 
}

int main(void) {
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int size = N * sizeof(int);

  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Allocate space for host copies of a, b, c and setup input values
  a = (int *)malloc(size);
  //random_ints(a, N);
  initialize(a, N);
  b = (int *)malloc(size);
  initialize(b, N);
  //random_ints(b, N);
  c = (int *)malloc(size);

  // * Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

  // Launch add() kernal on GPU with N blocks
  add<<<N,1>>>(d_a, d_b, d_c);

  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyHostToHost);

  // Cleanup, the functions used in cuda compiler is different
  free(a);
  free(b);
  free(c);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}
