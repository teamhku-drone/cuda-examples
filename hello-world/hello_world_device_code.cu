
#include <hip/hip_runtime.h>
#include<stdio.h>

// nvcc separate source code into device and host components

__global__ void mykernel(void) {
  // Device code is compiled by Nvidia compiler
  // This function is called from host code
}

int main(void) {
  // Host code goes here which is processed by standard host compiler
  // e.g. gcc

  // <<< >>> marks a call from host code to device code
  // a.k.a. kernal launch, return to the parameters (1,1)
  mykernel<<<1,1>>>();
  printf("Hello World!\n");
  return 0;
}
